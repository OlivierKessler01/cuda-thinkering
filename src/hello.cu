
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGpu(void)
{
    printf("Hello from GPU \n");
}

int main(void)
{
    printf("Hello from CPU \n");
    helloFromGpu <<<1, 10>>>();
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    hipDeviceReset();
    return 0;
}
