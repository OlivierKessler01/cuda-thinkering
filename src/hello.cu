
#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 * in_device_fn - This function call only be called from the device
 */
__device__ void in_device_fn() {
  printf("Hello from in_device_fn() in GPU thread %d\n", threadIdx.x);
}

/**
 * hello_from_gpu - This function can be called from device or host
 */
__global__ void hello_from_gpu(void) {
  printf("Hello from GPU thread %d\n", threadIdx.x);
  in_device_fn();
}

int main(void) {
  printf("Hello from CPU \n");
  hello_from_gpu<<<1, 100>>>();
  printf("CPU waits \n");

  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaerr));

  hipDeviceReset();
  return 0;
}
