
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGpu(void)
{
    printf("Hello from GPU \n");
}

int main(void)
{
    printf("Hello from CPU \n");
    helloFromGpu <<<1,10>>>();
    hipDeviceReset();
    return 0;
}
