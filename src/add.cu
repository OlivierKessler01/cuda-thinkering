
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(float *a, float *b, float *c, int N) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < N) {
    c[i] = a[i] + b[i];
  }
}

int main(void) {
  int N = 1000000000;

  size_t size = N * sizeof(float);
  float *a, *b, *c, *d_a, *d_b, *d_c;

  a = (float *)malloc(size);
  b = (float *)malloc(size);
  c = (float *)malloc(size);

  for (int i = 0; i < N; i++) {
    a[i] = (float)rand() / (float)(RAND_MAX / 100);
    b[i] = (float)rand() / (float)(RAND_MAX / 100);
  }

  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

  add<<<1000000, 1000>>>(d_a, d_b, d_c, N);

  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaerr));

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  return 0;
}
