
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(float *a, float *b, float *c, int N) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < N) {
    c[i] = a[i] + b[i];
  }
}

int main(void) {
  int N = 8;
  size_t size = N * sizeof(float);
  float *a, *b, *c, *d_a, *d_b, *d_c;

  a = (float *)malloc(size);
  b = (float *)malloc(size);
  c = (float *)malloc(size);

  for (int i = 0; i < N; i++) {
    a[i] = (float)rand() / (float)(RAND_MAX / 100);
    b[i] = (float)rand() / (float)(RAND_MAX / 100);
  }

  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

  add<<<2, 4>>>(d_a, d_b, d_c, N);

  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaerr));

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  printf("Results : \n");
  for (int i = 0; i < N; i++) {
    printf("%f + %f = %f \n", a[i], b[i], c[i]);
  }

  return 0;
}
