
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(float *a, float *b, float *c, int N) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < N) {
    c[i] = a[i] + b[i];
  }
}

int main(void) {
  int N = 1000000000;
  size_t size = N * sizeof(float);
  float *a, *b, *c, *d_a, *d_b;

  a = (float *)malloc(size);
  b = (float *)malloc(size);
  c = (float *)malloc(size);

  for (int i = 0; i < N; i++) {
    a[i] = (float)rand() / (float)(RAND_MAX / 100);
    b[i] = (float)rand() / (float)(RAND_MAX / 100);
  }

  for (int i = 0; i < N; i++) {
    c[i] = a[i] + b[i];
  }

  return 0;
}
