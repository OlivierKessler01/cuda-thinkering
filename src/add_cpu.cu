/**
 * Add a huge amount of float on a CPU
 * Made to be a comparison point for the same computation over GPUs (see add.cu)
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(float *a, float *b, float *c, int N) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < N) {
    c[i] = a[i] + b[i];
  }
}

int main(void) {
  int N = 1 <<26;
  size_t size = N * sizeof(float);
  float *a, *b, *c;

  a = (float *)malloc(size);
  b = (float *)malloc(size);
  c = (float *)malloc(size);

  for (int i = 0; i < N; i++) {
    a[i] = (float)rand() / (float)(RAND_MAX / 100);
    b[i] = (float)rand() / (float)(RAND_MAX / 100);
  }

  for (int i = 0; i < N; i++) {
    c[i] = a[i] + b[i];
  }

  return 0;
}
