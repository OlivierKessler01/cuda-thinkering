#include "hip/hip_runtime.h"
/**
 * Add a huge amount of float on a GPU
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__global__ void add(float *a, float *b, float *c, int N) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (i < N) {
    c[i] = a[i] + b[i];
  }
}

int main(void) {
  float *a, *b, *c, *d_a, *d_b, *d_c;

  int dev = 0;
  hipDeviceProp_t deviceProp;
  cudaGetDeviceProperties_v2(&deviceProp, dev);
  printf("Using device %d: %s\n", dev, deviceProp.name);
  hipSetDevice(dev);

  int nElem = 1 << 26;
  size_t size = nElem * sizeof(float);
  printf("Vector size %d\n", nElem);

  a = (float *)malloc(size);
  b = (float *)malloc(size);
  c = (float *)malloc(size);

  for (int i = 0; i < nElem; i++) {
    a[i] = (float)rand() / (float)(RAND_MAX / 100);
    b[i] = (float)rand() / (float)(RAND_MAX / 100);
  }

  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  int iLen = 1024;
  dim3 block(iLen);
  dim3 grid((nElem + block.x - 1) / block.x);

  add<<<grid, block>>>(d_a, d_b, d_c, nElem);

  printf("add<<%d,%d>>\n", grid.x, block.x);

  hipError_t cudaerr = hipDeviceSynchronize();

  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaerr));

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(a);
  free(b);
  free(c);

  return 0;
}
